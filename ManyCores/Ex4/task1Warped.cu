#include "hip/hip_runtime.h"
#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <cmath> //abs

#define BLOCKSIZE 256

/*__global__ void sumVectorKernel(const double* x, int N, double* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int warpIdx = threadIdx.x / warpSize;
    std::cout << "Warpsize: " << warpSize << std::endl;

    double sum = x[tid];

    // Warp shuffle reduction
    for (int i = warpSize; i > 0; i /= 2) {
        sum += __shfl_down_sync(-1, sum, i);
    }
     if (threadIdx.x % warpSize == 0) {atomicAdd(result, sum);}// thread 0 contains sum of all values within the warp
}*/


__global__ void sumVectorKernel(const double* x, int N, double* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int laneId = threadIdx.x % warpSize;
    //std::cout << "Warpsize: " << warpSize << std::endl;

    double sum = x[tid];

    // Warp shuffle reduction
    for (int i = warpSize / 2; i > 0; i /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, i, warpSize);
    }

    // The first thread in each warp adds its partial sum to the global result
    if (laneId == 0) {
        atomicAdd(result, sum);
    }
}

__global__ void OneNormKernel(const double* x, int N, double* result) {   //calculate one norm
    int tid = threadIdx.x;
    int warpIdx = threadIdx.x / warpSize;

   double sum = 0;
   for (int i = tid; i < N; i += blockDim.x * gridDim.x) {sum += std::abs(x[i]);}

    // Warp shuffle reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    // The first thread in each warp adds its partial sum to the global result
    if (threadIdx.x % warpSize == 0) {atomicAdd(result, sum);}
}


__global__ void TwoNormKernel(const double* x, int N, double* result) {   //calculate 2- norm
    int tid = threadIdx.x;
    int warpIdx = threadIdx.x / warpSize;

   double sum = 0;
   for (int i = tid; i < N; i += blockDim.x * gridDim.x) {sum += x[i] * x[i] ;}

    // Warp shuffle reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    // The first thread in each warp adds its partial sum to the global result
    if (threadIdx.x % warpSize == 0) {atomicAdd(result, sum);}
}

__global__ void ZeroCounterKernel(const double* x, int N, unsigned* result, double tol) {   //calc 0 entries==> smaller than tol
    int tid = threadIdx.x;
    int warpIdx = threadIdx.x / warpSize;

   unsigned numzeros = 0;
   for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        if( std::abs(x[i]) < tol )
            numzeros ++;
    }

    // Warp shuffle reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        numzeros += __shfl_down_sync(0xFFFFFFFF, numzeros, offset);
    }

    // The first thread in each warp adds its partial sum to the global result
    if (threadIdx.x % warpSize == 0) {atomicAdd(result, numzeros);}
}

int main(void){
  int N = 1'000'000;
     
    // Allocate and initialize arrays on CPU
  double *x = (double *)malloc(sizeof(double) * N);
  double alpha, oneNorm, twoNorm = 0;
  unsigned numzeros = 0;

  std::fill(x, x + 5000, 1);
  //std::fill( x + 5000, x + 5000 + 5000, -1);


  // Allocate and initialize arrays on GPU
  double *cuda_x;
  double *cuda_alpha, *cuda_OneNorm, *cuda_TwoNorm;
  unsigned *cuda_numzeros;
  
  CUDA_ERRCHK(hipMalloc(&cuda_x, sizeof(double) * N));
  CUDA_ERRCHK(hipMalloc(&cuda_alpha, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_OneNorm, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_TwoNorm, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_numzeros, sizeof(unsigned)));

  
  CUDA_ERRCHK(hipMemcpy(cuda_x, x, sizeof(double) * N, hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_OneNorm, &oneNorm, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_TwoNorm, &twoNorm, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_numzeros, &numzeros, sizeof(unsigned), hipMemcpyHostToDevice));

  
  sumVectorKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_alpha);
  OneNormKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_OneNorm);
  TwoNormKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_TwoNorm);
  ZeroCounterKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_numzeros, 1e-9);

  
  CUDA_ERRCHK(hipMemcpy(&alpha, cuda_alpha, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(&oneNorm, cuda_OneNorm, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(&twoNorm, cuda_TwoNorm, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(&numzeros, cuda_numzeros, sizeof(unsigned), hipMemcpyDeviceToHost));

  std::cout << "Result of summing entries: " << alpha << std::endl;
  std::cout << "Result of 1-NORM: " << oneNorm << std::endl;
  std::cout << "Result of 2-NORM: " << twoNorm << std::endl;
  std::cout << "Result of counting zero entries: " << numzeros << std::endl;

  // Clean up
  CUDA_ERRCHK(hipFree(cuda_x));
  CUDA_ERRCHK(hipFree(cuda_alpha));CUDA_ERRCHK(hipFree(cuda_OneNorm)); CUDA_ERRCHK(hipFree(cuda_TwoNorm)); CUDA_ERRCHK(hipFree(cuda_numzeros));

  free(x);

  return EXIT_SUCCESS;


}
