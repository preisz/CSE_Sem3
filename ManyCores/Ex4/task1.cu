#include "hip/hip_runtime.h"
#include "timer.hpp"
#include "cuda_errchk.hpp"
#include <algorithm>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <cmath> //abs

#define BLOCKSIZE 256

__global__ void initAll( double* cuda_alpha, double* cuda_oneNorm, double* cuda_twoNorm, unsigned* cudaNumZeros  ){
    *cuda_alpha = 0.0;
    *cuda_oneNorm = 0.0;
    *cuda_twoNorm = 0.0;
    *cudaNumZeros = 0;
}

__global__ void sumVectorKernel(const double* x, int N,  double* result) {    //calculate the sum of entries
    __shared__ double shared_mem[BLOCKSIZE];

    int localtid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

   double sum = 0;
   for (int i = gid; i < N; i += blockDim.x * gridDim.x) {sum += x[i]; }

  shared_mem[localtid] = sum;
  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    __syncthreads();
    if (threadIdx.x < k) {
      shared_mem[localtid] += shared_mem[localtid + k];
    }
  }

  if (threadIdx.x == 0) atomicAdd(result, shared_mem[0]);
}


__global__ void OneNormKernel(const double* x, int N, double* result) {   //calculate one norm
    __shared__ double shared_mem[BLOCKSIZE];

    int localtid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

   double sum = 0;
   for (int i = gid; i < N; i += blockDim.x * gridDim.x) {sum += std::abs(x[i]); }

  shared_mem[localtid] = sum;
  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    __syncthreads();
    if (threadIdx.x < k) {
      shared_mem[localtid] += shared_mem[localtid + k];
    }
  }

  if (threadIdx.x == 0) atomicAdd(result, shared_mem[0]);
}

__global__ void TwoNormKernel(const double* x, int N, double* result) {  //calculate two  norm
    __shared__ double shared_mem[BLOCKSIZE];

    int localtid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

   double sum = 0;
   for (int i = gid; i < N; i += blockDim.x * gridDim.x) {sum += x[i] * x[i] ; }

  shared_mem[localtid] = sum;
  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    __syncthreads();
    if (threadIdx.x < k) {
      shared_mem[localtid] += shared_mem[localtid + k];
    }
  }

  if (threadIdx.x == 0) atomicAdd(result, shared_mem[0]);
}


__global__ void ZeroCounterKernel(const double* x, int N, unsigned* result, double tol) {  //calculate 0 entries ==> smaller than tol
    __shared__ double shared_mem[BLOCKSIZE];

    int localtid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

   unsigned numZeros = 0;
   for (int i = gid; i < N; i += blockDim.x * gridDim.x) {
        if( std::abs(x[i])<tol ) 
            numZeros ++ ; 
    }

  shared_mem[localtid] = numZeros;
  for (int k = blockDim.x / 2; k > 0; k /= 2) {
    __syncthreads();
    if (threadIdx.x < k) {
      shared_mem[localtid] += shared_mem[localtid + k];
    }
  }

  if (threadIdx.x == 0) atomicAdd(result, shared_mem[0]);
}

int main(void){
  //int N = 1'000'000;
  Timer timer;
  std::vector<int> Nvals = { 100, 1'000, 10'000, 100'000, 1'000'000, 10'000'000, 50'000'000, 100'000'000, };
  std::cout << "****Using shared memory****\n";
  std::cout << "Length of vector N, Execution time for operations" << std::endl;

  for (int N : Nvals)   {
    // Allocate and initialize arrays on CPU
  double *x = (double *)malloc(sizeof(double) * N);
  double alpha = 0.0, oneNorm = 0.0, twoNorm = 0.0;
  unsigned numzeros = 0;

  int a = int(N/4);
  std::fill(x, x + a, 2);
  std::fill( x + a, x + (a + a + a), -1);


  // Allocate and initialize arrays on GPU
  double *cuda_x;
  double *cuda_alpha, *cuda_OneNorm, *cuda_TwoNorm;
  unsigned *cuda_numzeros;
  
  CUDA_ERRCHK(hipMalloc(&cuda_x, sizeof(double) * N));
  CUDA_ERRCHK(hipMalloc(&cuda_alpha, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_OneNorm, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_TwoNorm, sizeof(double)));
  CUDA_ERRCHK(hipMalloc(&cuda_numzeros, sizeof(unsigned)));

  
  CUDA_ERRCHK(hipMemcpy(cuda_x, x, sizeof(double) * N, hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_OneNorm, &oneNorm, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_TwoNorm, &twoNorm, sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cuda_numzeros, &numzeros, sizeof(unsigned), hipMemcpyHostToDevice));

  // execute functions and measure time
  CUDA_ERRCHK(hipDeviceSynchronize());   
  timer.reset();
    sumVectorKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_alpha);
    OneNormKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_OneNorm);
    TwoNormKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_TwoNorm);
    ZeroCounterKernel<<<BLOCKSIZE, BLOCKSIZE>>>(cuda_x, N, cuda_numzeros, 1e-9);
  CUDA_ERRCHK(hipDeviceSynchronize()); 
  double elapsed = timer.get(); // wait for kernel to finish, then print elapsed time

  std:: cout << N << "," << elapsed << std::endl;

  CUDA_ERRCHK(hipMemcpy(&alpha, cuda_alpha, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(&oneNorm, cuda_OneNorm, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(&twoNorm, cuda_TwoNorm, sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERRCHK(hipMemcpy(&numzeros, cuda_numzeros, sizeof(unsigned), hipMemcpyDeviceToHost));

/* DEBUG PART
  std::cout << "Result of summing entries: " << alpha << std::endl;
  std::cout << "Result of 1-NORM: " << oneNorm << std::endl;
  std::cout << "Result of 2-NORM: " << twoNorm << std::endl;
  std::cout << "Result of counting zero entries: " << numzeros << std::endl;
*/

  // Clean up
  CUDA_ERRCHK(hipFree(cuda_x));
  CUDA_ERRCHK(hipFree(cuda_alpha));CUDA_ERRCHK(hipFree(cuda_OneNorm)); CUDA_ERRCHK(hipFree(cuda_TwoNorm)); 
  CUDA_ERRCHK(hipFree(cuda_numzeros));

  free(x);

  }
  return EXIT_SUCCESS;

}